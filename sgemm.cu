#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

__global__ void naiveSgemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        float psum = 0.0;
        #pragma unroll
        for (int k = 0; k < K; k++) {
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

__global__ void mySgemmV1Aligned(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr]);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr]);

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < BK; k++) {
            #pragma unroll
            for (int m = 0; m < TM; m++) {
                #pragma unroll
                for (int n = 0; n < TN; n++) {
                    int comp_a_smem_m = ty * TM + m;
                    int comp_b_smem_n = tx * TN + n;
                    r_c[m][n] += s_a[comp_a_smem_m][k] * s_b[k][comp_b_smem_n];
                }
            }
        }

        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < TM; i++) {
        int store_c_gmem_m = by * BM + ty * TM + i;
        #pragma unroll
        for (int j = 0; j < TN; j += 4) {
            int store_c_gmem_n = bx * BN + tx * TN + j;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
        }
    }
}

__global__ void mySgemmV2Aligned(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BK][BM];
    __shared__ float s_b[BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];
    float r_comp_b[TN];
    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {

        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        s_a[load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();

        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[tk][ty * TM / 2         ]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[tk][tx * TN / 2         ]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[tk][tx * TN / 2 + BN / 2]);

            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}

__global__ void mySgemmV3Aligned(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[2][BK][BM];
    __shared__ float s_b[2][BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];
    float r_comp_b[TN];
    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    {
        int load_a_gmem_k = load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        s_a[0][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
    }

    __syncthreads();
    int smem_sel;
    int smem_sel_next;

    for (int bk = 1; bk < (K + BK - 1) / BK; bk++) {

        smem_sel = (bk - 1) & 1;
        smem_sel_next = bk & 1;

        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2         ]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2         ]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        s_a[smem_sel_next][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();
    }

    #pragma unroll
    for (int tk = 0; tk < BK; tk++) {
        FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel_next][tk][ty * TM / 2         ]);
        FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel_next][tk][ty * TM / 2 + BM / 2]);
        FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel_next][tk][tx * TN / 2         ]);
        FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel_next][tk][tx * TN / 2 + BN / 2]);

        #pragma unroll
        for (int tm = 0; tm < TM; tm++) {
            #pragma unroll
            for (int tn = 0; tn < TN; tn++) {
                r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
            }
        }
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}

float testMaxError(
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

float testCublasMaxError(const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    float cublas_alpha = 1.0;
    float cublas_beta = 0;
    // hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &cublas_alpha, d_a, K, d_b, N, &cublas_beta, d_c, M);
    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_b, N, d_a, K, &cublas_beta, d_c, N);

    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

float testPerformance(
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++)
        gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

float testCublasPerformance(const int M, const int N, const int K, const int repeat) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    float cublas_alpha = 1.0;
    float cublas_beta = 0;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        //hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &cublas_alpha, d_a, K, d_b, N, &cublas_beta, d_c, M);
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_b, N, d_a, K, &cublas_beta, d_c, N);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

int main() {

    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    // const int K_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};
    const int outer_repeat = 10, inner_repeat = 1;

    {
        printf("\nKernal = cublas\n");

        {
            const int M = 512, N = 512, K = 512;
            float max_error = testCublasMaxError(M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {
            const int TESTNUM = 15;

            for (int i = 0; i < TESTNUM; i++) {
                const int M = M_list[i], N = N_list[i], K = K_list[i];

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testCublasPerformance(M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    {
        printf("\nKernal = naiveSgemm\n");

        const int BM = 32, BN = 32;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) =
            naiveSgemm;

        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN, BM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {
            const int TESTNUM = 15;

            for (int i = 0; i < TESTNUM; i++) {
                const int M = M_list[i], N = N_list[i], K = K_list[i];

                dim3 blockDim(BN, BM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    {
        printf("\nKernal = mySgemmV1Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) =
            mySgemmV1Aligned;

        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {
            const int TESTNUM = 15;

            for (int i = 0; i < TESTNUM; i++) {
                const int M = M_list[i], N = N_list[i], K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }


    {
        printf("\nKernal = mySgemmV2Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) =
            mySgemmV2Aligned;

        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {
            const int TESTNUM = 15;

            for (int i = 0; i < TESTNUM; i++) {
                const int M = M_list[i], N = N_list[i], K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    {
        printf("\nKernal = mySgemmV3Aligned\n");

        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) =
            mySgemmV3Aligned;

        {
            const int M = 512, N = 512, K = 512;
            dim3 blockDim(BN / TN, BM / TM);
            dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
            float max_error = testMaxError(gpuSgemm, gridDim, blockDim, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        {
            const int TESTNUM = 15;

            for (int i = 0; i < TESTNUM; i++) {
                const int M = M_list[i], N = N_list[i], K = K_list[i];

                dim3 blockDim(BN / TN, BM / TM);
                dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

                double max_sec = 0.0;
                double min_sec = DBL_MAX;
                double total_sec = 0.0;

                for (int j = 0; j < outer_repeat; j++) {
                    double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
                    max_sec = max(max_sec, this_sec);
                    min_sec = min(min_sec, this_sec);
                    total_sec += this_sec;
                }

                double avg_sec = total_sec / outer_repeat;
                double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

                printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
            }
        }
    }

    return 0;
}
